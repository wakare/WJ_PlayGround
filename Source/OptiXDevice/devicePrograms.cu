#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>

#include "LaunchParams.h"

using namespace gdt;

namespace gdt {
    /*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
    extern "C" __constant__ LaunchParams optixLaunchParams;

    // for this simple example, we have a single ray type
    enum { SURFACE_RAY_TYPE=0, RAY_TYPE_COUNT };

    static __forceinline__ __device__
    void *unpackPointer( uint32_t i0, uint32_t i1 )
    {
        const uint64_t uptr = static_cast<uint64_t>( i0 ) << 32 | i1;
        void*           ptr = reinterpret_cast<void*>( uptr );
        return ptr;
    }

    static __forceinline__ __device__
    void  packPointer( void* ptr, uint32_t& i0, uint32_t& i1 )
    {
        const uint64_t uptr = reinterpret_cast<uint64_t>( ptr );
        i0 = uptr >> 32;
        i1 = uptr & 0x00000000ffffffff;
    }

    template<typename T>
    static __forceinline__ __device__ T *getPRD()
    {
        const uint32_t u0 = optixGetPayload_0();
        const uint32_t u1 = optixGetPayload_1();
        return reinterpret_cast<T*>( unpackPointer( u0, u1 ) );
    }

    static __forceinline__ __device__ vec3f colorMul(const vec3f& color0, const vec3f& color1)
    {
        return {color0.x * color1.x, color0.y * color1.y, color0.z * color1.z};
    }

    static __forceinline__ __device__ vec3f scaleVector(const vec3f& vector, float scale)
    {
        return {scale * vector.x,scale * vector.y,scale * vector.z };
    }

    static __forceinline__ __device__ bool hasNAN(const vec3f& vector)
    {
         return isnan(vector.x) || isnan(vector.y) || isnan(vector.z);
    }

    static __forceinline__ __device__ bool hasINF(const vec3f& vector)
    {
        return isinf(vector.x) || isinf(vector.y) || isinf(vector.z);
    }

    //------------------------------------------------------------------------------
    // closest hit and anyhit programs for radiance-type rays.
    //
    // Note eventually we will have to create one pair of those for each
    // ray type and each geometry type we want to render; but this
    // simple example doesn't use any actual geometries yet, so we only
    // create a single, dummy, set of them (we do have to have at least
    // one group of them to set up the SBT)
    //------------------------------------------------------------------------------

    extern "C" __global__ void __closesthit__radiance()
    {
        /*const int   primID = optixGetPrimitiveIndex();
        vec3f &prd = *(vec3f*)getPRD<vec3f>();
        prd = gdt::randomColor(primID);*/

        const HitGroupUserParams* hitParams = (const HitGroupUserParams*)optixGetSbtDataPointer();
        RayPayload& payload = (*getPRD<RayPayload>());
        ++payload.bounceCount;

        //printf("Current bounce count: %d\n", payload.bounceCount);

        const int   primID = optixGetPrimitiveIndex();
        const vec3i index = hitParams->index[primID];
        const vec3f& v0 = hitParams->vertex[index.x];
        const vec3f& v1 = hitParams->vertex[index.y];
        const vec3f& v2 = hitParams->vertex[index.z];
        const vec3f normal = normalize(cross(v1 - v0, v0 - v2));

        vec3f sampleDirection = vec3f(0.0f, 0.0f, 0.0f);
        float samplePdf = 0.0f;
        hitParams->MeshMaterial.SampleDirection(payload.RandGenerator, normal, optixGetWorldRayDirection(),
                                                sampleDirection, samplePdf);

        // Apply ray pdf
        float pdf = payload.pdf;
        //if (payload.bounceCount > 5)
        {
            if (payload.bounceCount > 20)
            {
                payload.color = hitParams->MeshMaterial.Emissive / pdf;
                return;
            }

            const vec3f& MatDiffuse = hitParams->MeshMaterial.Diffuse;
            float maxChannel = max(MatDiffuse.z, max(MatDiffuse.x, MatDiffuse.y));
            float rand = payload.RandGenerator();
            if (rand >= maxChannel || maxChannel < 0.00001f)
            {
                payload.color = hitParams->MeshMaterial.Emissive / pdf;
                return;
            }

            payload.pdf = maxChannel;
        }

        uint32_t u0 = optixGetPayload_0();
        uint32_t u1 = optixGetPayload_1();
        vec3f hitLocation = (vec3f)optixGetWorldRayOrigin() + scaleVector(optixGetWorldRayDirection(),
                                                                          optixGetRayTmax() - 0.00001f);

        //float2 baryCentricesCoord = optixGetTriangleBarycentrics();
        //vec3f location =  baryCentricesCoord.x * v1 + baryCentricesCoord.y * v2 + (1 - baryCentricesCoord.x - baryCentricesCoord.y) * v0;
        //printf ("location %f %f %f  bc %f %f %f  %f %f %f\n", hitLocation.x,hitLocation.y,hitLocation.z,
        //        baryCentricesCoord.x, baryCentricesCoord.y, 1 - baryCentricesCoord.x-baryCentricesCoord.y,
        //        location.x,location.y,location.z);

        optixTrace(optixLaunchParams.traversable,
                   hitLocation,
                   sampleDirection,
                   0.f,    // tmin
                   1e20f,  // tmax
                   0.0f,   // rayTime
                   OptixVisibilityMask( 255 ),
                   OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
                   SURFACE_RAY_TYPE,             // SBT offset
                   RAY_TYPE_COUNT,               // SBT stride
                   SURFACE_RAY_TYPE,             // missSBTIndex
                   u0, u1);

        payload.color = (colorMul(hitParams->MeshMaterial.Diffuse, payload.color) + hitParams->MeshMaterial.Emissive) / pdf;
    }

    extern "C" __global__ void __anyhit__radiance()
    { /*! for this simple example, this will remain empty */ }



    //------------------------------------------------------------------------------
    // miss program that gets called for any ray that did not have a
    // valid intersection
    //
    // as with the anyhit/closest hit programs, in this example we only
    // need to have _some_ dummy function to set up a valid SBT
    // ------------------------------------------------------------------------------

    extern "C" __global__ void __miss__radiance()
    {
        RayPayload &prd = *getPRD<RayPayload>();
        // set to constant white as background color
        if (prd.bounceCount == 0)
        {
            prd.color = {0.0f, 0.0f, 0.0f};
        }
    }

    //------------------------------------------------------------------------------
    // ray gen program - the actual rendering happens in here
    //------------------------------------------------------------------------------
    extern "C" __global__ void __raygen__renderFrame()
    {
        const int spp = optixLaunchParams.spp;

        // compute a test pattern based on pixel ID
        const int ix = optixGetLaunchIndex().x;
        const int iy = optixGetLaunchIndex().y;
        const int frameIndex = optixLaunchParams.frameIndex;

        const auto &camera = optixLaunchParams.camera;

        // normalized screen plane position, in [0,1]^2
        const vec2f screen(vec2f(ix+.5f, iy+.5f)
                           / vec2f(optixLaunchParams.frame.size));

        // generate ray direction
        vec3f rayDir = normalize(camera.direction
                                 + (screen.x - 0.5f) * camera.horizontal
                                 + (screen.y - 0.5f) * camera.vertical);

        // our per-ray data for this example. what we initialize it to
        // won't matter, since this value will be overwritten by either
        // the miss or hit program, anyway
        RayPayload Payload;
        Payload.RandGenerator = LCG<>((frameIndex * 21321424 * iy) ^ ix, (frameIndex * 92374894 * ix) ^ iy);

        //vec3f pixelColorPRD = vec3f(0.f);
        // the values we store the PRD pointer in:
        uint32_t u0, u1;
        //packPointer( &pixelColorPRD, u0, u1 );
        packPointer( &Payload, u0, u1 );

        vec3f totalColor = {0.0f, 0.0f, 0.0f};
        for (int i = 0; i < spp; ++i)
        {
            Payload.color = vec3f(0.0f, 0.0f, 0.0f);
            Payload.bounceCount = 0;
            Payload.pdf = 1.0f;

            optixTrace(optixLaunchParams.traversable,
                       camera.position,
                       rayDir,
                       0.f,    // tmin
                       1e20f,  // tmax
                       0.0f,   // rayTime
                       OptixVisibilityMask( 255 ),
                       OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
                       SURFACE_RAY_TYPE,             // SBT offset
                       RAY_TYPE_COUNT,               // SBT stride
                       SURFACE_RAY_TYPE,             // missSBTIndex
                       u0, u1 );
            totalColor += Payload.color;
        }

        totalColor /= spp;
        totalColor.x = clamp(totalColor.x, 0.0f, 1.0f);
        totalColor.y = clamp(totalColor.y, 0.0f, 1.0f);
        totalColor.z = clamp(totalColor.z, 0.0f, 1.0f);

        // and write to frame buffer ...
        const uint32_t fbIndex = ix+iy*optixLaunchParams.frame.size.x;

        // blend with previous frame result
        const float previousR = optixLaunchParams.sourceFrame.source[fbIndex].x;
        const float previousG = optixLaunchParams.sourceFrame.source[fbIndex].y;
        const float previousB = optixLaunchParams.sourceFrame.source[fbIndex].z;

        float alpha = (1.0f * frameIndex) / (frameIndex + 1);
        float oneMinusAlpha = 1.0f - alpha;
        float finalR = alpha * previousR + oneMinusAlpha * totalColor.x;
        float finalG = alpha * previousG + oneMinusAlpha * totalColor.y;
        float finalB = alpha * previousB + oneMinusAlpha * totalColor.z;

        optixLaunchParams.sourceFrame.source[fbIndex].x = finalR;
        optixLaunchParams.sourceFrame.source[fbIndex].y = finalG;
        optixLaunchParams.sourceFrame.source[fbIndex].z = finalB;

        const int finalRInt = int(255.99f * finalR);
        const int finalGInt = int(255.99f * finalG);
        const int finalBInt = int(255.99f * finalB);

        // convert to 32-bit rgba value (we explicitly set alpha to 0xff
        // to make stb_image_write happy ...
        const uint32_t rgba = 0xff000000
                              | (finalRInt<<0) | (finalGInt<<8) | (finalBInt<<16);

        optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
    }
  
} // ::osc
