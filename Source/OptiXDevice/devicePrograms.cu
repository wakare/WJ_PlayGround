#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2018-2019 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include <optix_device.h>

#include "LaunchParams.h"

using namespace gdt;

namespace gdt {
    /*! launch parameters in constant memory, filled in by optix upon
      optixLaunch (this gets filled in from the buffer we pass to
      optixLaunch) */
    extern "C" __constant__ LaunchParams optixLaunchParams;

    // for this simple example, we have a single ray type
    enum { SURFACE_RAY_TYPE=0, RAY_TYPE_COUNT };

    static __forceinline__ __device__
    void *unpackPointer( uint32_t i0, uint32_t i1 )
    {
        const uint64_t uptr = static_cast<uint64_t>( i0 ) << 32 | i1;
        void*           ptr = reinterpret_cast<void*>( uptr );
        return ptr;
    }

    static __forceinline__ __device__
    void  packPointer( void* ptr, uint32_t& i0, uint32_t& i1 )
    {
        const uint64_t uptr = reinterpret_cast<uint64_t>( ptr );
        i0 = uptr >> 32;
        i1 = uptr & 0x00000000ffffffff;
    }

    template<typename T>
    static __forceinline__ __device__ T *getPRD()
    {
        const uint32_t u0 = optixGetPayload_0();
        const uint32_t u1 = optixGetPayload_1();
        return reinterpret_cast<T*>( unpackPointer( u0, u1 ) );
    }

    static __forceinline__ __device__ vec3f colorMul(const vec3f& color0, const vec3f& color1)
    {
        return {color0.x * color1.x, color0.y * color1.y, color0.z * color1.z};
    }

    static __forceinline__ __device__ vec3f scaleVector(const vec3f& vector, float scale)
    {
        return {scale * vector.x,scale * vector.y,scale * vector.z };
    }

    //------------------------------------------------------------------------------
    // closest hit and anyhit programs for radiance-type rays.
    //
    // Note eventually we will have to create one pair of those for each
    // ray type and each geometry type we want to render; but this
    // simple example doesn't use any actual geometries yet, so we only
    // create a single, dummy, set of them (we do have to have at least
    // one group of them to set up the SBT)
    //------------------------------------------------------------------------------

    extern "C" __global__ void __closesthit__radiance()
    {
        /*const int   primID = optixGetPrimitiveIndex();
        vec3f &prd = *(vec3f*)getPRD<vec3f>();
        prd = gdt::randomColor(primID);*/

        const HitGroupUserParams* hitParams = (const HitGroupUserParams*)optixGetSbtDataPointer();
        RayPayload& payload = (*getPRD<RayPayload>());
        ++payload.bounceCount;

        // printf("Current bounce count: %d\n", payload.bounceCount);

        const int   primID = optixGetPrimitiveIndex();
        const vec3i index = hitParams->index[primID];
        const vec3f& v0 = hitParams->vertex[index.x];
        const vec3f& v1 = hitParams->vertex[index.y];
        const vec3f& v2 = hitParams->vertex[index.z];
        const vec3f normal = normalize(cross(v1 - v0, v0 - v2));

        vec3f sampleDirection = vec3f(0.0f, 0.0f, 0.0f);
        float samplePdf = 0.0f;
        hitParams->MeshMaterial.SampleDirection(payload.RandGenerator, normal, optixGetWorldRayDirection(),
                                                sampleDirection, samplePdf);

        // Apply ray pdf
        payload.color /= payload.pdf;

        if (payload.bounceCount > 5)
        {
            return;
            float maxChannel = max(payload.color.z, max(payload.color.x, payload.color.y));
            float rand = payload.RandGenerator();
            if (rand < maxChannel)
            {
                return;
            }

            payload.pdf = rand;
        }

        uint32_t u0 = optixGetPayload_0();
        uint32_t u1 = optixGetPayload_1();

        const float t = optixGetRayTmax() - 0.00001f;
        vec3f hitLocation = (vec3f)optixGetWorldRayOrigin() + scaleVector(optixGetWorldRayDirection(), t);

        optixTrace(optixLaunchParams.traversable,
                   hitLocation,
                   sampleDirection,
                   0.f,    // tmin
                   1e20f,  // tmax
                   0.0f,   // rayTime
                   OptixVisibilityMask( 255 ),
                   OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
                   SURFACE_RAY_TYPE,             // SBT offset
                   RAY_TYPE_COUNT,               // SBT stride
                   SURFACE_RAY_TYPE,             // missSBTIndex
                   u0, u1);

        payload.color = colorMul(hitParams->MeshMaterial.Diffuse, payload.color) + hitParams->MeshMaterial.Emissive;
    }

    extern "C" __global__ void __anyhit__radiance()
    { /*! for this simple example, this will remain empty */ }



    //------------------------------------------------------------------------------
    // miss program that gets called for any ray that did not have a
    // valid intersection
    //
    // as with the anyhit/closest hit programs, in this example we only
    // need to have _some_ dummy function to set up a valid SBT
    // ------------------------------------------------------------------------------

    extern "C" __global__ void __miss__radiance()
    {
        RayPayload &prd = *getPRD<RayPayload>();
        // set to constant white as background color
        if (prd.bounceCount == 0)
        {
            prd.color = {0.0f, 0.0f, 0.0f};
        }
    }

    //------------------------------------------------------------------------------
    // ray gen program - the actual rendering happens in here
    //------------------------------------------------------------------------------
    extern "C" __global__ void __raygen__renderFrame()
    {
        const int spp = 16;

        // compute a test pattern based on pixel ID
        const int ix = optixGetLaunchIndex().x;
        const int iy = optixGetLaunchIndex().y;

        const auto &camera = optixLaunchParams.camera;

        // normalized screen plane position, in [0,1]^2
        const vec2f screen(vec2f(ix+.5f,iy+.5f)
                           / vec2f(optixLaunchParams.frame.size));

        // generate ray direction
        vec3f rayDir = normalize(camera.direction
                                 + (screen.x - 0.5f) * camera.horizontal
                                 + (screen.y - 0.5f) * camera.vertical);

        // our per-ray data for this example. what we initialize it to
        // won't matter, since this value will be overwritten by either
        // the miss or hit program, anyway
        RayPayload Payload;
        Payload.RandGenerator = LCG<>(ix,iy);

        //vec3f pixelColorPRD = vec3f(0.f);
        // the values we store the PRD pointer in:
        uint32_t u0, u1;
        //packPointer( &pixelColorPRD, u0, u1 );
        packPointer( &Payload, u0, u1 );

        vec3f totalColor = {0.0f, 0.0f, 0.0f};
        for (int i = 0; i < spp; ++i)
        {
            Payload.color = vec3f(0.0f, 0.0f, 0.0f);
            Payload.bounceCount = 0;
            Payload.pdf = 1.0f;

            optixTrace(optixLaunchParams.traversable,
                       camera.position,
                       rayDir,
                       0.f,    // tmin
                       1e20f,  // tmax
                       0.0f,   // rayTime
                       OptixVisibilityMask( 255 ),
                       OPTIX_RAY_FLAG_DISABLE_ANYHIT,//OPTIX_RAY_FLAG_NONE,
                       SURFACE_RAY_TYPE,             // SBT offset
                       RAY_TYPE_COUNT,               // SBT stride
                       SURFACE_RAY_TYPE,             // missSBTIndex
                       u0, u1 );
            totalColor += Payload.color;
        }

        totalColor /= spp;

        const int r = int(255.99f*totalColor.x);
        const int g = int(255.99f*totalColor.y);
        const int b = int(255.99f*totalColor.z);

        // convert to 32-bit rgba value (we explicitly set alpha to 0xff
        // to make stb_image_write happy ...
        const uint32_t rgba = 0xff000000
                              | (r<<0) | (g<<8) | (b<<16);

        // and write to frame buffer ...
        const uint32_t fbIndex = ix+iy*optixLaunchParams.frame.size.x;
        optixLaunchParams.frame.colorBuffer[fbIndex] = rgba;
    }
  
} // ::osc
